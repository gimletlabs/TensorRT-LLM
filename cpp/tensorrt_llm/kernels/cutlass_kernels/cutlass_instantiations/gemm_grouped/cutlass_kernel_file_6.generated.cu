#include "hip/hip_runtime.h"
#include "tensorrt_llm/kernels/cutlass_kernels/moe_gemm/launchers/moe_gemm_tma_ws_launcher.inl"
namespace tensorrt_llm
{
namespace kernels
{
namespace cutlass_kernels
{


#if 1

        INSTANTIATE_TMA_WARP_SPECIALIZED_MOE_GEMM(Sm90, half, half, half,
                EpilogueOpDefault, NONE, 128, 16, 64, 1, 1, 1, false);

#endif


#if 1

        INSTANTIATE_TMA_WARP_SPECIALIZED_MOE_GEMM(Sm90, half, half, half,
                EpilogueOpDefault, NONE, 128, 16, 64, 2, 1, 1, false);

#endif


#if 1

        INSTANTIATE_TMA_WARP_SPECIALIZED_MOE_GEMM(Sm90, half, half, half,
                EpilogueOpDefault, NONE, 128, 32, 64, 1, 1, 1, false);

#endif


#if 1

        INSTANTIATE_TMA_WARP_SPECIALIZED_MOE_GEMM(Sm90, half, half, half,
                EpilogueOpDefault, NONE, 128, 32, 64, 2, 1, 1, false);

#endif


#if 1

        INSTANTIATE_TMA_WARP_SPECIALIZED_MOE_GEMM(Sm90, half, half, half,
                EpilogueOpDefault, NONE, 128, 64, 64, 1, 1, 1, false);

#endif


#if 1

        INSTANTIATE_TMA_WARP_SPECIALIZED_MOE_GEMM(Sm90, half, half, half,
                EpilogueOpDefault, NONE, 128, 64, 64, 2, 1, 1, false);

#endif


#if 1

        INSTANTIATE_TMA_WARP_SPECIALIZED_MOE_GEMM(Sm90, half, half, half,
                EpilogueOpDefault, NONE, 128, 128, 64, 1, 1, 1, false);

#endif


#if 1

        INSTANTIATE_TMA_WARP_SPECIALIZED_MOE_GEMM(Sm90, half, half, half,
                EpilogueOpDefault, NONE, 128, 128, 64, 1, 2, 1, false);

#endif


#if 1

        INSTANTIATE_TMA_WARP_SPECIALIZED_MOE_GEMM(Sm90, half, half, half,
                EpilogueOpDefault, NONE, 128, 128, 64, 2, 1, 1, false);

#endif


#if 1

        INSTANTIATE_TMA_WARP_SPECIALIZED_MOE_GEMM(Sm90, half, half, half,
                EpilogueOpDefault, NONE, 128, 128, 64, 2, 2, 1, false);

#endif


#if 1

        INSTANTIATE_TMA_WARP_SPECIALIZED_MOE_GEMM(Sm90, half, half, half,
                EpilogueOpDefault, NONE, 128, 256, 64, 1, 1, 1, false);

#endif


#if 1

        INSTANTIATE_TMA_WARP_SPECIALIZED_MOE_GEMM(Sm90, half, half, half,
                EpilogueOpDefault, NONE, 128, 256, 64, 1, 2, 1, false);

#endif


#if 1

        INSTANTIATE_TMA_WARP_SPECIALIZED_MOE_GEMM(Sm90, half, half, half,
                EpilogueOpDefault, NONE, 128, 256, 64, 2, 1, 1, false);

#endif


#if 1

        INSTANTIATE_TMA_WARP_SPECIALIZED_MOE_GEMM(Sm90, half, half, half,
                EpilogueOpDefault, NONE, 128, 256, 64, 2, 2, 1, false);

#endif


#if 1

        INSTANTIATE_TMA_WARP_SPECIALIZED_MOE_GEMM(Sm90, half, half, half,
                EpilogueOpDefault, FINALIZE, 128, 16, 64, 1, 1, 1, false);

#endif


#if 1

        INSTANTIATE_TMA_WARP_SPECIALIZED_MOE_GEMM(Sm90, half, half, half,
                EpilogueOpDefault, FINALIZE, 128, 16, 64, 2, 1, 1, false);

#endif


#if 1

        INSTANTIATE_TMA_WARP_SPECIALIZED_MOE_GEMM(Sm90, half, half, half,
                EpilogueOpDefault, FINALIZE, 128, 32, 64, 1, 1, 1, false);

#endif


#if 1

        INSTANTIATE_TMA_WARP_SPECIALIZED_MOE_GEMM(Sm90, half, half, half,
                EpilogueOpDefault, FINALIZE, 128, 32, 64, 2, 1, 1, false);

#endif


#if 1

        INSTANTIATE_TMA_WARP_SPECIALIZED_MOE_GEMM(Sm90, half, half, half,
                EpilogueOpDefault, FINALIZE, 128, 64, 64, 1, 1, 1, false);

#endif


#if 1

        INSTANTIATE_TMA_WARP_SPECIALIZED_MOE_GEMM(Sm90, half, half, half,
                EpilogueOpDefault, FINALIZE, 128, 64, 64, 2, 1, 1, false);

#endif


#if 1

        INSTANTIATE_TMA_WARP_SPECIALIZED_MOE_GEMM(Sm90, half, half, half,
                EpilogueOpDefault, FINALIZE, 128, 128, 64, 1, 1, 1, false);

#endif


#if 1

        INSTANTIATE_TMA_WARP_SPECIALIZED_MOE_GEMM(Sm90, half, half, half,
                EpilogueOpDefault, FINALIZE, 128, 128, 64, 1, 2, 1, false);

#endif


#if 1

        INSTANTIATE_TMA_WARP_SPECIALIZED_MOE_GEMM(Sm90, half, half, half,
                EpilogueOpDefault, FINALIZE, 128, 128, 64, 2, 1, 1, false);

#endif


#if 1

        INSTANTIATE_TMA_WARP_SPECIALIZED_MOE_GEMM(Sm90, half, half, half,
                EpilogueOpDefault, FINALIZE, 128, 128, 64, 2, 2, 1, false);

#endif


#if 1

        INSTANTIATE_TMA_WARP_SPECIALIZED_MOE_GEMM(Sm90, half, half, half,
                EpilogueOpDefault, FINALIZE, 128, 256, 64, 1, 1, 1, false);

#endif


#if 1

        INSTANTIATE_TMA_WARP_SPECIALIZED_MOE_GEMM(Sm90, half, half, half,
                EpilogueOpDefault, FINALIZE, 128, 256, 64, 1, 2, 1, false);

#endif


#if 1

        INSTANTIATE_TMA_WARP_SPECIALIZED_MOE_GEMM(Sm90, half, half, half,
                EpilogueOpDefault, FINALIZE, 128, 256, 64, 2, 1, 1, false);

#endif


#if 1

        INSTANTIATE_TMA_WARP_SPECIALIZED_MOE_GEMM(Sm90, half, half, half,
                EpilogueOpDefault, FINALIZE, 128, 256, 64, 2, 2, 1, false);

#endif


#if defined(ENABLE_BF16)

        INSTANTIATE_TMA_WARP_SPECIALIZED_MOE_GEMM(Sm90, __hip_bfloat16, __hip_bfloat16, __hip_bfloat16,
                EpilogueOpDefault, NONE, 128, 16, 64, 1, 1, 1, false);

#endif


#if defined(ENABLE_BF16)

        INSTANTIATE_TMA_WARP_SPECIALIZED_MOE_GEMM(Sm90, __hip_bfloat16, __hip_bfloat16, __hip_bfloat16,
                EpilogueOpDefault, NONE, 128, 16, 64, 2, 1, 1, false);

#endif


#if defined(ENABLE_BF16)

        INSTANTIATE_TMA_WARP_SPECIALIZED_MOE_GEMM(Sm90, __hip_bfloat16, __hip_bfloat16, __hip_bfloat16,
                EpilogueOpDefault, NONE, 128, 32, 64, 1, 1, 1, false);

#endif


#if defined(ENABLE_BF16)

        INSTANTIATE_TMA_WARP_SPECIALIZED_MOE_GEMM(Sm90, __hip_bfloat16, __hip_bfloat16, __hip_bfloat16,
                EpilogueOpDefault, NONE, 128, 32, 64, 2, 1, 1, false);

#endif


#if defined(ENABLE_BF16)

        INSTANTIATE_TMA_WARP_SPECIALIZED_MOE_GEMM(Sm90, __hip_bfloat16, __hip_bfloat16, __hip_bfloat16,
                EpilogueOpDefault, NONE, 128, 64, 64, 1, 1, 1, false);

#endif


#if defined(ENABLE_BF16)

        INSTANTIATE_TMA_WARP_SPECIALIZED_MOE_GEMM(Sm90, __hip_bfloat16, __hip_bfloat16, __hip_bfloat16,
                EpilogueOpDefault, NONE, 128, 64, 64, 2, 1, 1, false);

#endif


#if defined(ENABLE_BF16)

        INSTANTIATE_TMA_WARP_SPECIALIZED_MOE_GEMM(Sm90, __hip_bfloat16, __hip_bfloat16, __hip_bfloat16,
                EpilogueOpDefault, NONE, 128, 128, 64, 1, 1, 1, false);

#endif


#if defined(ENABLE_BF16)

        INSTANTIATE_TMA_WARP_SPECIALIZED_MOE_GEMM(Sm90, __hip_bfloat16, __hip_bfloat16, __hip_bfloat16,
                EpilogueOpDefault, NONE, 128, 128, 64, 1, 2, 1, false);

#endif


#if defined(ENABLE_BF16)

        INSTANTIATE_TMA_WARP_SPECIALIZED_MOE_GEMM(Sm90, __hip_bfloat16, __hip_bfloat16, __hip_bfloat16,
                EpilogueOpDefault, NONE, 128, 128, 64, 2, 1, 1, false);

#endif


#if defined(ENABLE_BF16)

        INSTANTIATE_TMA_WARP_SPECIALIZED_MOE_GEMM(Sm90, __hip_bfloat16, __hip_bfloat16, __hip_bfloat16,
                EpilogueOpDefault, NONE, 128, 128, 64, 2, 2, 1, false);

#endif


#if defined(ENABLE_BF16)

        INSTANTIATE_TMA_WARP_SPECIALIZED_MOE_GEMM(Sm90, __hip_bfloat16, __hip_bfloat16, __hip_bfloat16,
                EpilogueOpDefault, NONE, 128, 256, 64, 1, 1, 1, false);

#endif


#if defined(ENABLE_BF16)

        INSTANTIATE_TMA_WARP_SPECIALIZED_MOE_GEMM(Sm90, __hip_bfloat16, __hip_bfloat16, __hip_bfloat16,
                EpilogueOpDefault, NONE, 128, 256, 64, 1, 2, 1, false);

#endif


#if defined(ENABLE_BF16)

        INSTANTIATE_TMA_WARP_SPECIALIZED_MOE_GEMM(Sm90, __hip_bfloat16, __hip_bfloat16, __hip_bfloat16,
                EpilogueOpDefault, NONE, 128, 256, 64, 2, 1, 1, false);

#endif


#if defined(ENABLE_BF16)

        INSTANTIATE_TMA_WARP_SPECIALIZED_MOE_GEMM(Sm90, __hip_bfloat16, __hip_bfloat16, __hip_bfloat16,
                EpilogueOpDefault, NONE, 128, 256, 64, 2, 2, 1, false);

#endif


#if defined(ENABLE_BF16)

        INSTANTIATE_TMA_WARP_SPECIALIZED_MOE_GEMM(Sm90, __hip_bfloat16, __hip_bfloat16, __hip_bfloat16,
                EpilogueOpDefault, FINALIZE, 128, 16, 64, 1, 1, 1, false);

#endif


#if defined(ENABLE_BF16)

        INSTANTIATE_TMA_WARP_SPECIALIZED_MOE_GEMM(Sm90, __hip_bfloat16, __hip_bfloat16, __hip_bfloat16,
                EpilogueOpDefault, FINALIZE, 128, 16, 64, 2, 1, 1, false);

#endif


#if defined(ENABLE_BF16)

        INSTANTIATE_TMA_WARP_SPECIALIZED_MOE_GEMM(Sm90, __hip_bfloat16, __hip_bfloat16, __hip_bfloat16,
                EpilogueOpDefault, FINALIZE, 128, 32, 64, 1, 1, 1, false);

#endif


#if defined(ENABLE_BF16)

        INSTANTIATE_TMA_WARP_SPECIALIZED_MOE_GEMM(Sm90, __hip_bfloat16, __hip_bfloat16, __hip_bfloat16,
                EpilogueOpDefault, FINALIZE, 128, 32, 64, 2, 1, 1, false);

#endif


#if defined(ENABLE_BF16)

        INSTANTIATE_TMA_WARP_SPECIALIZED_MOE_GEMM(Sm90, __hip_bfloat16, __hip_bfloat16, __hip_bfloat16,
                EpilogueOpDefault, FINALIZE, 128, 64, 64, 1, 1, 1, false);

#endif


#if defined(ENABLE_BF16)

        INSTANTIATE_TMA_WARP_SPECIALIZED_MOE_GEMM(Sm90, __hip_bfloat16, __hip_bfloat16, __hip_bfloat16,
                EpilogueOpDefault, FINALIZE, 128, 64, 64, 2, 1, 1, false);

#endif


#if defined(ENABLE_BF16)

        INSTANTIATE_TMA_WARP_SPECIALIZED_MOE_GEMM(Sm90, __hip_bfloat16, __hip_bfloat16, __hip_bfloat16,
                EpilogueOpDefault, FINALIZE, 128, 128, 64, 1, 1, 1, false);

#endif


#if defined(ENABLE_BF16)

        INSTANTIATE_TMA_WARP_SPECIALIZED_MOE_GEMM(Sm90, __hip_bfloat16, __hip_bfloat16, __hip_bfloat16,
                EpilogueOpDefault, FINALIZE, 128, 128, 64, 1, 2, 1, false);

#endif


#if defined(ENABLE_BF16)

        INSTANTIATE_TMA_WARP_SPECIALIZED_MOE_GEMM(Sm90, __hip_bfloat16, __hip_bfloat16, __hip_bfloat16,
                EpilogueOpDefault, FINALIZE, 128, 128, 64, 2, 1, 1, false);

#endif


#if defined(ENABLE_BF16)

        INSTANTIATE_TMA_WARP_SPECIALIZED_MOE_GEMM(Sm90, __hip_bfloat16, __hip_bfloat16, __hip_bfloat16,
                EpilogueOpDefault, FINALIZE, 128, 128, 64, 2, 2, 1, false);

#endif


#if defined(ENABLE_BF16)

        INSTANTIATE_TMA_WARP_SPECIALIZED_MOE_GEMM(Sm90, __hip_bfloat16, __hip_bfloat16, __hip_bfloat16,
                EpilogueOpDefault, FINALIZE, 128, 256, 64, 1, 1, 1, false);

#endif


#if defined(ENABLE_BF16)

        INSTANTIATE_TMA_WARP_SPECIALIZED_MOE_GEMM(Sm90, __hip_bfloat16, __hip_bfloat16, __hip_bfloat16,
                EpilogueOpDefault, FINALIZE, 128, 256, 64, 1, 2, 1, false);

#endif


#if defined(ENABLE_BF16)

        INSTANTIATE_TMA_WARP_SPECIALIZED_MOE_GEMM(Sm90, __hip_bfloat16, __hip_bfloat16, __hip_bfloat16,
                EpilogueOpDefault, FINALIZE, 128, 256, 64, 2, 1, 1, false);

#endif


#if defined(ENABLE_BF16)

        INSTANTIATE_TMA_WARP_SPECIALIZED_MOE_GEMM(Sm90, __hip_bfloat16, __hip_bfloat16, __hip_bfloat16,
                EpilogueOpDefault, FINALIZE, 128, 256, 64, 2, 2, 1, false);

#endif


#if 1

        INSTANTIATE_TMA_WARP_SPECIALIZED_MOE_GEMM(Sm90, float, float, float,
                EpilogueOpDefault, NONE, 128, 16, 32, 1, 1, 1, false);

#endif


#if 1

        INSTANTIATE_TMA_WARP_SPECIALIZED_MOE_GEMM(Sm90, float, float, float,
                EpilogueOpDefault, NONE, 128, 16, 32, 2, 1, 1, false);

#endif


#if 1

        INSTANTIATE_TMA_WARP_SPECIALIZED_MOE_GEMM(Sm90, float, float, float,
                EpilogueOpDefault, NONE, 128, 32, 32, 1, 1, 1, false);

#endif


#if 1

        INSTANTIATE_TMA_WARP_SPECIALIZED_MOE_GEMM(Sm90, float, float, float,
                EpilogueOpDefault, NONE, 128, 32, 32, 2, 1, 1, false);

#endif


#if 1

        INSTANTIATE_TMA_WARP_SPECIALIZED_MOE_GEMM(Sm90, float, float, float,
                EpilogueOpDefault, NONE, 128, 64, 32, 1, 1, 1, false);

#endif


#if 1

        INSTANTIATE_TMA_WARP_SPECIALIZED_MOE_GEMM(Sm90, float, float, float,
                EpilogueOpDefault, NONE, 128, 64, 32, 2, 1, 1, false);

#endif


#if 1

        INSTANTIATE_TMA_WARP_SPECIALIZED_MOE_GEMM(Sm90, float, float, float,
                EpilogueOpDefault, NONE, 128, 128, 32, 1, 1, 1, false);

#endif


#if 1

        INSTANTIATE_TMA_WARP_SPECIALIZED_MOE_GEMM(Sm90, float, float, float,
                EpilogueOpDefault, NONE, 128, 128, 32, 1, 2, 1, false);

#endif


#if 1

        INSTANTIATE_TMA_WARP_SPECIALIZED_MOE_GEMM(Sm90, float, float, float,
                EpilogueOpDefault, NONE, 128, 128, 32, 2, 1, 1, false);

#endif


#if 1

        INSTANTIATE_TMA_WARP_SPECIALIZED_MOE_GEMM(Sm90, float, float, float,
                EpilogueOpDefault, NONE, 128, 128, 32, 2, 2, 1, false);

#endif


#if 1

        INSTANTIATE_TMA_WARP_SPECIALIZED_MOE_GEMM(Sm90, float, float, float,
                EpilogueOpDefault, NONE, 128, 256, 32, 1, 1, 1, false);

#endif


#if 1

        INSTANTIATE_TMA_WARP_SPECIALIZED_MOE_GEMM(Sm90, float, float, float,
                EpilogueOpDefault, NONE, 128, 256, 32, 1, 2, 1, false);

#endif


#if 1

        INSTANTIATE_TMA_WARP_SPECIALIZED_MOE_GEMM(Sm90, float, float, float,
                EpilogueOpDefault, NONE, 128, 256, 32, 2, 1, 1, false);

#endif


#if 1

        INSTANTIATE_TMA_WARP_SPECIALIZED_MOE_GEMM(Sm90, float, float, float,
                EpilogueOpDefault, NONE, 128, 256, 32, 2, 2, 1, false);

#endif


#if 1

        INSTANTIATE_TMA_WARP_SPECIALIZED_MOE_GEMM(Sm90, float, float, float,
                EpilogueOpDefault, FINALIZE, 128, 16, 32, 1, 1, 1, false);

#endif


#if 1

        INSTANTIATE_TMA_WARP_SPECIALIZED_MOE_GEMM(Sm90, float, float, float,
                EpilogueOpDefault, FINALIZE, 128, 16, 32, 2, 1, 1, false);

#endif


#if 1

        INSTANTIATE_TMA_WARP_SPECIALIZED_MOE_GEMM(Sm90, float, float, float,
                EpilogueOpDefault, FINALIZE, 128, 32, 32, 1, 1, 1, false);

#endif


#if 1

        INSTANTIATE_TMA_WARP_SPECIALIZED_MOE_GEMM(Sm90, float, float, float,
                EpilogueOpDefault, FINALIZE, 128, 32, 32, 2, 1, 1, false);

#endif


#if 1

        INSTANTIATE_TMA_WARP_SPECIALIZED_MOE_GEMM(Sm90, float, float, float,
                EpilogueOpDefault, FINALIZE, 128, 64, 32, 1, 1, 1, false);

#endif


#if 1

        INSTANTIATE_TMA_WARP_SPECIALIZED_MOE_GEMM(Sm90, float, float, float,
                EpilogueOpDefault, FINALIZE, 128, 64, 32, 2, 1, 1, false);

#endif


#if 1

        INSTANTIATE_TMA_WARP_SPECIALIZED_MOE_GEMM(Sm90, float, float, float,
                EpilogueOpDefault, FINALIZE, 128, 128, 32, 1, 1, 1, false);

#endif


#if 1

        INSTANTIATE_TMA_WARP_SPECIALIZED_MOE_GEMM(Sm90, float, float, float,
                EpilogueOpDefault, FINALIZE, 128, 128, 32, 1, 2, 1, false);

#endif


#if 1

        INSTANTIATE_TMA_WARP_SPECIALIZED_MOE_GEMM(Sm90, float, float, float,
                EpilogueOpDefault, FINALIZE, 128, 128, 32, 2, 1, 1, false);

#endif


#if 1

        INSTANTIATE_TMA_WARP_SPECIALIZED_MOE_GEMM(Sm90, float, float, float,
                EpilogueOpDefault, FINALIZE, 128, 128, 32, 2, 2, 1, false);

#endif


#if 1

        INSTANTIATE_TMA_WARP_SPECIALIZED_MOE_GEMM(Sm90, float, float, float,
                EpilogueOpDefault, FINALIZE, 128, 256, 32, 1, 1, 1, false);

#endif


#if 1

        INSTANTIATE_TMA_WARP_SPECIALIZED_MOE_GEMM(Sm90, float, float, float,
                EpilogueOpDefault, FINALIZE, 128, 256, 32, 1, 2, 1, false);

#endif


#if 1

        INSTANTIATE_TMA_WARP_SPECIALIZED_MOE_GEMM(Sm90, float, float, float,
                EpilogueOpDefault, FINALIZE, 128, 256, 32, 2, 1, 1, false);

#endif


#if 1

        INSTANTIATE_TMA_WARP_SPECIALIZED_MOE_GEMM(Sm90, float, float, float,
                EpilogueOpDefault, FINALIZE, 128, 256, 32, 2, 2, 1, false);

#endif


#if defined(ENABLE_FP8)

        INSTANTIATE_TMA_WARP_SPECIALIZED_MOE_GEMM(Sm90, __hip_fp8_e4m3_fnuz, __hip_fp8_e4m3_fnuz, half,
                EpilogueOpDefault, NONE, 128, 16, 128, 1, 1, 1, false);

#endif


#if defined(ENABLE_FP8)

        INSTANTIATE_TMA_WARP_SPECIALIZED_MOE_GEMM(Sm90, __hip_fp8_e4m3_fnuz, __hip_fp8_e4m3_fnuz, __hip_bfloat16,
                EpilogueOpDefault, NONE, 128, 16, 128, 1, 1, 1, false);

#endif


#if defined(ENABLE_FP8)

        INSTANTIATE_TMA_WARP_SPECIALIZED_MOE_GEMM(Sm90, __hip_fp8_e4m3_fnuz, __hip_fp8_e4m3_fnuz, half,
                EpilogueOpDefault, NONE, 128, 16, 128, 2, 1, 1, false);

#endif


#if defined(ENABLE_FP8)

        INSTANTIATE_TMA_WARP_SPECIALIZED_MOE_GEMM(Sm90, __hip_fp8_e4m3_fnuz, __hip_fp8_e4m3_fnuz, __hip_bfloat16,
                EpilogueOpDefault, NONE, 128, 16, 128, 2, 1, 1, false);

#endif


#if defined(ENABLE_FP8)

        INSTANTIATE_TMA_WARP_SPECIALIZED_MOE_GEMM(Sm90, __hip_fp8_e4m3_fnuz, __hip_fp8_e4m3_fnuz, half,
                EpilogueOpDefault, NONE, 128, 32, 128, 1, 1, 1, false);

#endif


#if defined(ENABLE_FP8)

        INSTANTIATE_TMA_WARP_SPECIALIZED_MOE_GEMM(Sm90, __hip_fp8_e4m3_fnuz, __hip_fp8_e4m3_fnuz, __hip_bfloat16,
                EpilogueOpDefault, NONE, 128, 32, 128, 1, 1, 1, false);

#endif


#if defined(ENABLE_FP8)

        INSTANTIATE_TMA_WARP_SPECIALIZED_MOE_GEMM(Sm90, __hip_fp8_e4m3_fnuz, __hip_fp8_e4m3_fnuz, half,
                EpilogueOpDefault, NONE, 128, 32, 128, 2, 1, 1, false);

#endif


#if defined(ENABLE_FP8)

        INSTANTIATE_TMA_WARP_SPECIALIZED_MOE_GEMM(Sm90, __hip_fp8_e4m3_fnuz, __hip_fp8_e4m3_fnuz, __hip_bfloat16,
                EpilogueOpDefault, NONE, 128, 32, 128, 2, 1, 1, false);

#endif


#if defined(ENABLE_FP8)

        INSTANTIATE_TMA_WARP_SPECIALIZED_MOE_GEMM(Sm90, __hip_fp8_e4m3_fnuz, __hip_fp8_e4m3_fnuz, half,
                EpilogueOpDefault, NONE, 128, 64, 128, 1, 1, 1, false);

#endif


#if defined(ENABLE_FP8)

        INSTANTIATE_TMA_WARP_SPECIALIZED_MOE_GEMM(Sm90, __hip_fp8_e4m3_fnuz, __hip_fp8_e4m3_fnuz, __hip_bfloat16,
                EpilogueOpDefault, NONE, 128, 64, 128, 1, 1, 1, false);

#endif


#if defined(ENABLE_FP8)

        INSTANTIATE_TMA_WARP_SPECIALIZED_MOE_GEMM(Sm90, __hip_fp8_e4m3_fnuz, __hip_fp8_e4m3_fnuz, half,
                EpilogueOpDefault, NONE, 128, 64, 128, 2, 1, 1, false);

#endif


#if defined(ENABLE_FP8)

        INSTANTIATE_TMA_WARP_SPECIALIZED_MOE_GEMM(Sm90, __hip_fp8_e4m3_fnuz, __hip_fp8_e4m3_fnuz, __hip_bfloat16,
                EpilogueOpDefault, NONE, 128, 64, 128, 2, 1, 1, false);

#endif


#if defined(ENABLE_FP8)

        INSTANTIATE_TMA_WARP_SPECIALIZED_MOE_GEMM(Sm90, __hip_fp8_e4m3_fnuz, __hip_fp8_e4m3_fnuz, half,
                EpilogueOpDefault, NONE, 128, 128, 128, 1, 1, 1, false);

#endif


#if defined(ENABLE_FP8)

        INSTANTIATE_TMA_WARP_SPECIALIZED_MOE_GEMM(Sm90, __hip_fp8_e4m3_fnuz, __hip_fp8_e4m3_fnuz, __hip_bfloat16,
                EpilogueOpDefault, NONE, 128, 128, 128, 1, 1, 1, false);

#endif


#if defined(ENABLE_FP8)

        INSTANTIATE_TMA_WARP_SPECIALIZED_MOE_GEMM(Sm90, __hip_fp8_e4m3_fnuz, __hip_fp8_e4m3_fnuz, half,
                EpilogueOpDefault, NONE, 128, 128, 128, 1, 2, 1, false);

#endif


#if defined(ENABLE_FP8)

        INSTANTIATE_TMA_WARP_SPECIALIZED_MOE_GEMM(Sm90, __hip_fp8_e4m3_fnuz, __hip_fp8_e4m3_fnuz, __hip_bfloat16,
                EpilogueOpDefault, NONE, 128, 128, 128, 1, 2, 1, false);

#endif


#if defined(ENABLE_FP8)

        INSTANTIATE_TMA_WARP_SPECIALIZED_MOE_GEMM(Sm90, __hip_fp8_e4m3_fnuz, __hip_fp8_e4m3_fnuz, half,
                EpilogueOpDefault, NONE, 128, 128, 128, 2, 1, 1, false);

#endif


#if defined(ENABLE_FP8)

        INSTANTIATE_TMA_WARP_SPECIALIZED_MOE_GEMM(Sm90, __hip_fp8_e4m3_fnuz, __hip_fp8_e4m3_fnuz, __hip_bfloat16,
                EpilogueOpDefault, NONE, 128, 128, 128, 2, 1, 1, false);

#endif


#if defined(ENABLE_FP8)

        INSTANTIATE_TMA_WARP_SPECIALIZED_MOE_GEMM(Sm90, __hip_fp8_e4m3_fnuz, __hip_fp8_e4m3_fnuz, half,
                EpilogueOpDefault, NONE, 128, 128, 128, 2, 2, 1, false);

#endif


#if defined(ENABLE_FP8)

        INSTANTIATE_TMA_WARP_SPECIALIZED_MOE_GEMM(Sm90, __hip_fp8_e4m3_fnuz, __hip_fp8_e4m3_fnuz, __hip_bfloat16,
                EpilogueOpDefault, NONE, 128, 128, 128, 2, 2, 1, false);

#endif


#if defined(ENABLE_FP8)

        INSTANTIATE_TMA_WARP_SPECIALIZED_MOE_GEMM(Sm90, __hip_fp8_e4m3_fnuz, __hip_fp8_e4m3_fnuz, half,
                EpilogueOpDefault, NONE, 128, 256, 128, 1, 1, 1, false);

#endif


#if defined(ENABLE_FP8)

        INSTANTIATE_TMA_WARP_SPECIALIZED_MOE_GEMM(Sm90, __hip_fp8_e4m3_fnuz, __hip_fp8_e4m3_fnuz, __hip_bfloat16,
                EpilogueOpDefault, NONE, 128, 256, 128, 1, 1, 1, false);

#endif


#if defined(ENABLE_FP8)

        INSTANTIATE_TMA_WARP_SPECIALIZED_MOE_GEMM(Sm90, __hip_fp8_e4m3_fnuz, __hip_fp8_e4m3_fnuz, half,
                EpilogueOpDefault, NONE, 128, 256, 128, 1, 2, 1, false);

#endif


#if defined(ENABLE_FP8)

        INSTANTIATE_TMA_WARP_SPECIALIZED_MOE_GEMM(Sm90, __hip_fp8_e4m3_fnuz, __hip_fp8_e4m3_fnuz, __hip_bfloat16,
                EpilogueOpDefault, NONE, 128, 256, 128, 1, 2, 1, false);

#endif


#if defined(ENABLE_FP8)

        INSTANTIATE_TMA_WARP_SPECIALIZED_MOE_GEMM(Sm90, __hip_fp8_e4m3_fnuz, __hip_fp8_e4m3_fnuz, half,
                EpilogueOpDefault, NONE, 128, 256, 128, 2, 1, 1, false);

#endif


#if defined(ENABLE_FP8)

        INSTANTIATE_TMA_WARP_SPECIALIZED_MOE_GEMM(Sm90, __hip_fp8_e4m3_fnuz, __hip_fp8_e4m3_fnuz, __hip_bfloat16,
                EpilogueOpDefault, NONE, 128, 256, 128, 2, 1, 1, false);

#endif


#if defined(ENABLE_FP8)

        INSTANTIATE_TMA_WARP_SPECIALIZED_MOE_GEMM(Sm90, __hip_fp8_e4m3_fnuz, __hip_fp8_e4m3_fnuz, half,
                EpilogueOpDefault, NONE, 128, 256, 128, 2, 2, 1, false);

#endif


#if defined(ENABLE_FP8)

        INSTANTIATE_TMA_WARP_SPECIALIZED_MOE_GEMM(Sm90, __hip_fp8_e4m3_fnuz, __hip_fp8_e4m3_fnuz, __hip_bfloat16,
                EpilogueOpDefault, NONE, 128, 256, 128, 2, 2, 1, false);

#endif


#if defined(ENABLE_FP8)

        INSTANTIATE_TMA_WARP_SPECIALIZED_MOE_GEMM(Sm90, __hip_fp8_e4m3_fnuz, __hip_fp8_e4m3_fnuz, half,
                EpilogueOpDefault, FINALIZE, 128, 16, 128, 1, 1, 1, false);

#endif


#if defined(ENABLE_FP8)

        INSTANTIATE_TMA_WARP_SPECIALIZED_MOE_GEMM(Sm90, __hip_fp8_e4m3_fnuz, __hip_fp8_e4m3_fnuz, __hip_bfloat16,
                EpilogueOpDefault, FINALIZE, 128, 16, 128, 1, 1, 1, false);

#endif


#if defined(ENABLE_FP8)

        INSTANTIATE_TMA_WARP_SPECIALIZED_MOE_GEMM(Sm90, __hip_fp8_e4m3_fnuz, __hip_fp8_e4m3_fnuz, half,
                EpilogueOpDefault, FINALIZE, 128, 16, 128, 2, 1, 1, false);

#endif


#if defined(ENABLE_FP8)

        INSTANTIATE_TMA_WARP_SPECIALIZED_MOE_GEMM(Sm90, __hip_fp8_e4m3_fnuz, __hip_fp8_e4m3_fnuz, __hip_bfloat16,
                EpilogueOpDefault, FINALIZE, 128, 16, 128, 2, 1, 1, false);

#endif


#if defined(ENABLE_FP8)

        INSTANTIATE_TMA_WARP_SPECIALIZED_MOE_GEMM(Sm90, __hip_fp8_e4m3_fnuz, __hip_fp8_e4m3_fnuz, half,
                EpilogueOpDefault, FINALIZE, 128, 32, 128, 1, 1, 1, false);

#endif


#if defined(ENABLE_FP8)

        INSTANTIATE_TMA_WARP_SPECIALIZED_MOE_GEMM(Sm90, __hip_fp8_e4m3_fnuz, __hip_fp8_e4m3_fnuz, __hip_bfloat16,
                EpilogueOpDefault, FINALIZE, 128, 32, 128, 1, 1, 1, false);

#endif


#if defined(ENABLE_FP8)

        INSTANTIATE_TMA_WARP_SPECIALIZED_MOE_GEMM(Sm90, __hip_fp8_e4m3_fnuz, __hip_fp8_e4m3_fnuz, half,
                EpilogueOpDefault, FINALIZE, 128, 32, 128, 2, 1, 1, false);

#endif


#if defined(ENABLE_FP8)

        INSTANTIATE_TMA_WARP_SPECIALIZED_MOE_GEMM(Sm90, __hip_fp8_e4m3_fnuz, __hip_fp8_e4m3_fnuz, __hip_bfloat16,
                EpilogueOpDefault, FINALIZE, 128, 32, 128, 2, 1, 1, false);

#endif


#if defined(ENABLE_FP8)

        INSTANTIATE_TMA_WARP_SPECIALIZED_MOE_GEMM(Sm90, __hip_fp8_e4m3_fnuz, __hip_fp8_e4m3_fnuz, half,
                EpilogueOpDefault, FINALIZE, 128, 64, 128, 1, 1, 1, false);

#endif


#if defined(ENABLE_FP8)

        INSTANTIATE_TMA_WARP_SPECIALIZED_MOE_GEMM(Sm90, __hip_fp8_e4m3_fnuz, __hip_fp8_e4m3_fnuz, __hip_bfloat16,
                EpilogueOpDefault, FINALIZE, 128, 64, 128, 1, 1, 1, false);

#endif


#if defined(ENABLE_FP8)

        INSTANTIATE_TMA_WARP_SPECIALIZED_MOE_GEMM(Sm90, __hip_fp8_e4m3_fnuz, __hip_fp8_e4m3_fnuz, half,
                EpilogueOpDefault, FINALIZE, 128, 64, 128, 2, 1, 1, false);

#endif


#if defined(ENABLE_FP8)

        INSTANTIATE_TMA_WARP_SPECIALIZED_MOE_GEMM(Sm90, __hip_fp8_e4m3_fnuz, __hip_fp8_e4m3_fnuz, __hip_bfloat16,
                EpilogueOpDefault, FINALIZE, 128, 64, 128, 2, 1, 1, false);

#endif


#if defined(ENABLE_FP8)

        INSTANTIATE_TMA_WARP_SPECIALIZED_MOE_GEMM(Sm90, __hip_fp8_e4m3_fnuz, __hip_fp8_e4m3_fnuz, half,
                EpilogueOpDefault, FINALIZE, 128, 128, 128, 1, 1, 1, false);

#endif


#if defined(ENABLE_FP8)

        INSTANTIATE_TMA_WARP_SPECIALIZED_MOE_GEMM(Sm90, __hip_fp8_e4m3_fnuz, __hip_fp8_e4m3_fnuz, __hip_bfloat16,
                EpilogueOpDefault, FINALIZE, 128, 128, 128, 1, 1, 1, false);

#endif


#if defined(ENABLE_FP8)

        INSTANTIATE_TMA_WARP_SPECIALIZED_MOE_GEMM(Sm90, __hip_fp8_e4m3_fnuz, __hip_fp8_e4m3_fnuz, half,
                EpilogueOpDefault, FINALIZE, 128, 128, 128, 1, 2, 1, false);

#endif


#if defined(ENABLE_FP8)

        INSTANTIATE_TMA_WARP_SPECIALIZED_MOE_GEMM(Sm90, __hip_fp8_e4m3_fnuz, __hip_fp8_e4m3_fnuz, __hip_bfloat16,
                EpilogueOpDefault, FINALIZE, 128, 128, 128, 1, 2, 1, false);

#endif


#if defined(ENABLE_FP8)

        INSTANTIATE_TMA_WARP_SPECIALIZED_MOE_GEMM(Sm90, __hip_fp8_e4m3_fnuz, __hip_fp8_e4m3_fnuz, half,
                EpilogueOpDefault, FINALIZE, 128, 128, 128, 2, 1, 1, false);

#endif


#if defined(ENABLE_FP8)

        INSTANTIATE_TMA_WARP_SPECIALIZED_MOE_GEMM(Sm90, __hip_fp8_e4m3_fnuz, __hip_fp8_e4m3_fnuz, __hip_bfloat16,
                EpilogueOpDefault, FINALIZE, 128, 128, 128, 2, 1, 1, false);

#endif


#if defined(ENABLE_FP8)

        INSTANTIATE_TMA_WARP_SPECIALIZED_MOE_GEMM(Sm90, __hip_fp8_e4m3_fnuz, __hip_fp8_e4m3_fnuz, half,
                EpilogueOpDefault, FINALIZE, 128, 128, 128, 2, 2, 1, false);

#endif


#if defined(ENABLE_FP8)

        INSTANTIATE_TMA_WARP_SPECIALIZED_MOE_GEMM(Sm90, __hip_fp8_e4m3_fnuz, __hip_fp8_e4m3_fnuz, __hip_bfloat16,
                EpilogueOpDefault, FINALIZE, 128, 128, 128, 2, 2, 1, false);

#endif


#if defined(ENABLE_FP8)

        INSTANTIATE_TMA_WARP_SPECIALIZED_MOE_GEMM(Sm90, __hip_fp8_e4m3_fnuz, __hip_fp8_e4m3_fnuz, half,
                EpilogueOpDefault, FINALIZE, 128, 256, 128, 1, 1, 1, false);

#endif


#if defined(ENABLE_FP8)

        INSTANTIATE_TMA_WARP_SPECIALIZED_MOE_GEMM(Sm90, __hip_fp8_e4m3_fnuz, __hip_fp8_e4m3_fnuz, __hip_bfloat16,
                EpilogueOpDefault, FINALIZE, 128, 256, 128, 1, 1, 1, false);

#endif


#if defined(ENABLE_FP8)

        INSTANTIATE_TMA_WARP_SPECIALIZED_MOE_GEMM(Sm90, __hip_fp8_e4m3_fnuz, __hip_fp8_e4m3_fnuz, half,
                EpilogueOpDefault, FINALIZE, 128, 256, 128, 1, 2, 1, false);

#endif


#if defined(ENABLE_FP8)

        INSTANTIATE_TMA_WARP_SPECIALIZED_MOE_GEMM(Sm90, __hip_fp8_e4m3_fnuz, __hip_fp8_e4m3_fnuz, __hip_bfloat16,
                EpilogueOpDefault, FINALIZE, 128, 256, 128, 1, 2, 1, false);

#endif


#if defined(ENABLE_FP8)

        INSTANTIATE_TMA_WARP_SPECIALIZED_MOE_GEMM(Sm90, __hip_fp8_e4m3_fnuz, __hip_fp8_e4m3_fnuz, half,
                EpilogueOpDefault, FINALIZE, 128, 256, 128, 2, 1, 1, false);

#endif


#if defined(ENABLE_FP8)

        INSTANTIATE_TMA_WARP_SPECIALIZED_MOE_GEMM(Sm90, __hip_fp8_e4m3_fnuz, __hip_fp8_e4m3_fnuz, __hip_bfloat16,
                EpilogueOpDefault, FINALIZE, 128, 256, 128, 2, 1, 1, false);

#endif


#if defined(ENABLE_FP8)

        INSTANTIATE_TMA_WARP_SPECIALIZED_MOE_GEMM(Sm90, __hip_fp8_e4m3_fnuz, __hip_fp8_e4m3_fnuz, half,
                EpilogueOpDefault, FINALIZE, 128, 256, 128, 2, 2, 1, false);

#endif


#if defined(ENABLE_FP8)

        INSTANTIATE_TMA_WARP_SPECIALIZED_MOE_GEMM(Sm90, __hip_fp8_e4m3_fnuz, __hip_fp8_e4m3_fnuz, __hip_bfloat16,
                EpilogueOpDefault, FINALIZE, 128, 256, 128, 2, 2, 1, false);

#endif


} // namespace cutlass_kernels
} // namespace kernels
} // namespace tensorrt_llm
